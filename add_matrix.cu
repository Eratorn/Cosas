
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <assert.h>
#define N 10000
#define MAT_GPU 1 << 2
#define MAT_CPU 1 << 3

__device__ void set_element(int *matrix, int value, int cols, int x, int y)
{
    matrix[x + cols * y] = value;
}
__device__ int get_element(int *matrix, int cols, int x, int y)
{
    return matrix[x + cols * y];
}

__global__ void add(int *a, int *b, int *c, int rows, int cols)
{
    //----------                                      dimension
    int x = blockIdx.x * blockDim.x + threadIdx.x; //[0 0 0 0 0] [0 0 0 0 0] <- bloque
    int y = blockIdx.y * blockDim.y + threadIdx.y; // |_ threadIdx.x
    if (x < cols && y < rows)
    {
        int a_e = get_element(a, cols, x, y);
        int b_e = get_element(b, cols, x, y);
        set_element(c, a_e + b_e, cols, x, y);
    }
}

int *allocate_matrix(int kind, int rows, int cols)
{
    int *matrix = NULL;
    if (kind == MAT_GPU)
    {
        hipMalloc((void **)&matrix, sizeof(int) * rows * cols);
    }
    else if (kind == MAT_CPU)
    {
        matrix = (int *)malloc(sizeof(int) * rows * cols);
    }
    return matrix;
}
void random_fill_matrix(int *matrix, int rows, int cols)
{
    for (int idx = 0; idx < rows * cols; idx++)
    {
        matrix[idx] = rand() % (1 << 8);
    }
}
int main()
{
    int *a_host_matrix = allocate_matrix(MAT_CPU, N, N);
    int *b_host_matrix = allocate_matrix(MAT_CPU, N, N);
    int *result_host_matrix = allocate_matrix(MAT_CPU, N, N);

    int *a_device_matrix = allocate_matrix(MAT_GPU, N, N);
    int *b_device_matrix = allocate_matrix(MAT_GPU, N, N);
    int *result_device_matrix = allocate_matrix(MAT_GPU, N, N);

    random_fill_matrix(a_host_matrix, N, N);
    random_fill_matrix(b_host_matrix, N, N);
    hipMemcpy((void *)a_device_matrix, (void *)a_host_matrix, sizeof(int) * N * N, hipMemcpyHostToDevice);
    hipMemcpy((void *)b_device_matrix, (void *)b_host_matrix, sizeof(int) * N * N, hipMemcpyHostToDevice);
    dim3 bdim(int(ceil(N / 32.0f)), int(ceil(N / 32.0f))); //la estructura de los hilos es en forma de grid, donde cada celda es una matriz de 32x32
    dim3 tdim(32, 32);
    clock_t start = clock();
    add<<<bdim, tdim>>>(a_device_matrix, b_device_matrix, result_device_matrix, N, N);
    hipDeviceSynchronize();
    hipMemcpy((void *)result_host_matrix, (void *)result_device_matrix, sizeof(int) * N * N, hipMemcpyDeviceToHost);
    clock_t end = clock();
    printf("tiempo de ejecucion: %f\n", double(end - start) / CLOCKS_PER_SEC);
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            assert(result_host_matrix[j + i * N] == a_host_matrix[j + i * N] + b_host_matrix[j + i * N]);
        }
    }

    return 0;
}
