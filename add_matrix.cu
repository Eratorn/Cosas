
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <assert.h>
#define N 1000
#define MAT_GPU 1 << 2
#define MAT_CPU 1 << 3

__device__ void set_element(int *matrix, int value, int cols, int x, int y)
{
    matrix[x + cols * y] = value;
}
__device__ int get_element(int *matrix, int cols, int x, int y)
{
    return matrix[x + cols * y];
}

__global__ void add(int *a, int *b, int *c, int rows, int cols)
{
    //----------                                      dimension
    int x = blockIdx.x * blockDim.x + threadIdx.x; //[0 0 0 0 0] [0 0 0 0 0] <- bloque
    int y = blockIdx.y * blockDim.y + threadIdx.y; // |-> threadIdx.x
    if (x < cols && y < rows)
    {
        int a_e = get_element(a, cols, x, y);
        int b_e = get_element(b, cols, x, y);
        set_element(c, a_e + b_e, cols, x, y);
    }
}
__global__ void mult(int *a, int *b, int *c, int n)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n && j < n)
    {
        int res = 0;
        for (int k = 0; k < n; k++)
        {
            res += a[k + i * n] * b[j + k * n];
        }
        c[j + i * n] = res;
    }
}
int *allocate_matrix(int kind, int rows, int cols)
{
    int *matrix = NULL;
    if (kind == MAT_GPU)
    {
        hipMalloc((void **)&matrix, sizeof(int) * rows * cols);
    }
    else if (kind == MAT_CPU)
    {
        matrix = (int *)malloc(sizeof(int) * rows * cols);
    }
    return matrix;
}
void random_fill_matrix(int *matrix, int rows, int cols)
{
    for (int idx = 0; idx < rows * cols; idx++)
    {
        matrix[idx] = rand() % 10;
    }
}
void cpu_mul(int *a, int *b, int *c, int n)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            int res = 0;

            for (int k = 0; k < n; k++)
            {
                res += a[k + i * n] * b[j + k * n];
            }
            c[j + i * n] = res;
        }
    }
}
void print_matrix(int *m, int rows, int cols)
{
    for (int i = 0; i < rows; i++)

    {
        for (int j = 0; j < cols; j++)

        {
            printf("%d ", m[j + i * cols]);
        }
        printf("\n");
    }
    printf("-----------------------------------\n");
}
int main()
{
    srand(time(nullptr));
    int *a_host_matrix = allocate_matrix(MAT_CPU, N, N);
    int *b_host_matrix = allocate_matrix(MAT_CPU, N, N);
    int *result_host_matrix = allocate_matrix(MAT_CPU, N, N);

    int *a_device_matrix = allocate_matrix(MAT_GPU, N, N);
    int *b_device_matrix = allocate_matrix(MAT_GPU, N, N);
    int *result_device_matrix = allocate_matrix(MAT_GPU, N, N);

    random_fill_matrix(a_host_matrix, N, N);
    random_fill_matrix(b_host_matrix, N, N);
    //print_matrix(a_host_matrix, N, N);
    //print_matrix(b_host_matrix, N, N);
    //cpu_mul(a_host_matrix, b_host_matrix, result_host_matrix, N);
    //print_matrix(result_host_matrix, N, N);
    hipMemcpy((void *)a_device_matrix, (void *)a_host_matrix, sizeof(int) * N * N, hipMemcpyHostToDevice);
    hipMemcpy((void *)b_device_matrix, (void *)b_host_matrix, sizeof(int) * N * N, hipMemcpyHostToDevice);
    dim3 bdim(int(ceil(N / 32.0f)), int(ceil(N / 32.0f))); //la estructura de los hilos es en forma de grid, donde cada celda es una matriz de 32x32
    dim3 tdim(32, 32);
    clock_t start = clock();
    mult<<<bdim, tdim>>>(a_device_matrix, b_device_matrix, result_device_matrix, N);
    hipDeviceSynchronize();
    hipMemcpy((void *)result_host_matrix, (void *)result_device_matrix, sizeof(int) * N * N, hipMemcpyDeviceToHost);
    clock_t end = clock();
    printf("tiempo de ejecucion: %f\n", double(end - start) / CLOCKS_PER_SEC);
    //print_matrix(result_host_matrix, N, N);
    return 0;
}
